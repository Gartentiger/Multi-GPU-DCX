#include "hip/hip_runtime.h"
#include <mpi.h>
#include <stdio.h>
#include <stdlib.h>
#include <err.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <iostream>

__global__ void printFirst(int* array, int rank)
{
    printf("Thread idx: %d, Rank: %d, array[0]: %d\n", threadIdx.x, rank, array[0]);
}

int main(int argc, char** argv)
{
    MPI_Init(&argc, &argv);

    // Get the size of the group associated with communicator MPI_COMM_WORLD
    int world_size;
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);

    // Get the rank of the calling process in the communicator MPI_COMM_WORLD
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    int world_rank;
    MPI_Comm_rank(MPI_COMM_WORLD, &world_rank);
    printf("* WorldSize: %d, Rank: %d Cuda devices: %d\n", world_size, world_rank, deviceCount);
    // std::vector<int> input2(2u * comm.size(), comm.rank_signed());
    // std::vector<int> output = comm.alltoall(send_buf(input2));
    // printf("Rank: %d, Size: %d, SRank: %d\n", comm.rank(), output[0], output[1]);
    printf("* Allocate memory [%d],GPU\n", world_rank);
    int* d_a;
    if (hipMalloc((void**)&d_a, 100 * sizeof(int)) != hipSuccess)
    {
        auto error = hipGetErrorString(hipGetLastError());
        // errx(1, "hipMalloc d_a[] failed");
        printf("Error malloc %d, %s\n", world_rank, error);
        return 1;
    }
    hipMemset(d_a, 0, 100 * sizeof(int));
    printFirst << <1, 1 >> > (d_a, world_rank);
    int err = 0;
    MPI_Status status;
    // From [1],GPU to [0],GPU
    if (world_rank == 1)
    {
        hipMemset(d_a, 1, 100 * sizeof(int));
        printf("Memset %d \n", world_rank);
        printFirst << <1, 1 >> > (d_a, world_rank);
        err = MPI_Send(d_a, 100, MPI_INT, 0, 2, MPI_COMM_WORLD);
        printf("* Send from [%d] GPU\n", world_rank);
    }
    else if (world_rank == 0)
    {
        err = MPI_Recv(d_a, 100, MPI_INT, 1, 2, MPI_COMM_WORLD, &status);
        printf("* Receive to [%d] GPU\n", world_rank);
        printFirst << <1, 1 >> > (d_a, world_rank);
    }
    if (err != MPI_SUCCESS)
    {
        // errx(2, "MPI transport from [1],GPU to [0],GPU failed");
        printf("Error transport");
        return 1;
    }
    printf("* Free memory on [%d],GPU\n", world_rank);
    hipFree(d_a);

    // Terminates MPI execution environment
    MPI_Finalize();
    return 0;
}