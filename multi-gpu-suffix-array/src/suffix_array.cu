#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> // For syntax completion

#include <cstdio>
#include <cassert>
#include <array>
#include <cmath>

#include "io.cuh"

#include "stages.h"
#include "suffixarrayperformancemeasurements.hpp"

#include "suffix_array_kernels.cuh"
#include "suffixarraymemorymanager.hpp"
#include "cuda_helpers.h"
#include "remerge/remergemanager.hpp"
#include "remerge/remerge_gpu_topology_helper.hpp"

#include "gossip/all_to_all.cuh"
#include "gossip/multisplit.cuh"
#include "distrib_merge/distrib_merge.hpp"

#include <chrono>
#include <numeric>
#include <random>
#include <thread>
#include <vector>

#include <mpi.h>

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <kamping/checking_casts.hpp>
#include <kamping/collectives/alltoall.hpp>
#include <kamping/data_buffer.hpp>
#include <kamping/environment.hpp>
#include <kamping/measurements/printer.hpp>
#include <kamping/measurements/timer.hpp>
#include <kamping/named_parameters.hpp>
#include <kamping/communicator.hpp>
#include <kamping/p2p/recv.hpp>
#include <kamping/p2p/send.hpp>
#include <nvToolsExt.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

static const uint NUM_GPUS = 4;

#ifdef DGX1_TOPOLOGY
#include "gossip/all_to_all_dgx1.cuh"
static_assert(NUM_GPUS == 8, "DGX-1 topology can only be used with 8 GPUs");
template <size_t NUM_GPUS>
using All2All = gossip::All2AllDGX1<NUM_GPUS>;
template <size_t NUM_GPUS, class mtypes>
using ReMergeTopology = crossGPUReMerge::DGX1TopologyHelper<NUM_GPUS, mtypes>;
template <typename key_t, typename value_t, typename index_t, size_t NUM_GPUS>
using DistribMergeTopology = distrib_merge::DGX1TopologyHelper<key_t, value_t, index_t, NUM_GPUS>;
#else
#include "gossip/all_to_all.cuh"
// static_assert(NUM_GPUS <= 4, "At the moment, there is no node with more than 4 all-connected nodes. This is likely a configuration error.");

template <size_t NUM_GPUS>
using All2All = gossip::All2All<NUM_GPUS>;
template <size_t NUM_GPUS, class mtypes>
using ReMergeTopology = crossGPUReMerge::MergeGPUAllConnectedTopologyHelper<NUM_GPUS, mtypes>;
template <typename key_t, typename value_t, typename index_t, size_t NUM_GPUS>
using DistribMergeTopology = distrib_merge::DistribMergeAllConnectedTopologyHelper<key_t, value_t, index_t, NUM_GPUS>;
#endif

#if defined(__HIPCC__)
#define _KLC_SIMPLE_(num_elements, stream) <<<std::min(MAX_GRID_SIZE, SDIV((num_elements), BLOCK_SIZE)), BLOCK_SIZE, 0, (stream)>>>
#define _KLC_SIMPLE_ITEMS_PER_THREAD_(num_elements, items_per_thread, stream) <<<std::min(MAX_GRID_SIZE, SDIV((num_elements), BLOCK_SIZE*(items_per_thread))), BLOCK_SIZE, 0, (stream)>>>
#define _KLC_(...) <<<__VA_ARGS__>>>
#else
#define __forceinline__
#define _KLC_SIMPLE_(num_elements, stream)
#define _KLC_SIMPLE_ITEMS_PER_THREAD_(num_elements, items_per_thread, stream)
#define _KLC_(...)
#endif

struct S12PartitioningFunctor : public std::unary_function<sa_index_t, uint32_t>
{
    sa_index_t split_divisor;
    uint max_v;

    __forceinline__
        S12PartitioningFunctor(sa_index_t split_divisor_, uint max_v_)
        : split_divisor(split_divisor_), max_v(max_v_)
    {
    }

    __host__ __device__ __forceinline__ uint32_t operator()(sa_index_t x) const
    {
        return min(((x - 1) / split_divisor), max_v);
    }
};

struct S0Comparator : public std::binary_function<MergeStageSuffixS0HalfKey, MergeStageSuffixS0HalfKey, bool>
{
    __host__ __device__ __forceinline__ bool operator()(const MergeStageSuffixS0HalfKey& a, const MergeStageSuffixS0HalfKey& b) const
    {
        if (a.chars[0] == b.chars[0])
            return a.rank_p1 < b.rank_p1;
        else
            return a.chars[0] < b.chars[0];
    }
};

struct MergeCompFunctor : std::binary_function<MergeStageSuffix, MergeStageSuffix, bool>
{
    __host__ __device__ __forceinline__ bool operator()(const MergeStageSuffix& a, const MergeStageSuffix& b) const
    {
        if (a.index % 3 == 0)
        {
            assert(b.index % 3 != 0);
            if (b.index % 3 == 1)
            {
                if (a.chars[0] == b.chars[0])
                    return a.rank_p1 < b.rank_p1;
                return a.chars[0] < b.chars[0];
            }
            else
            {
                if (a.chars[0] == b.chars[0])
                {
                    if (a.chars[1] == b.chars[1])
                    {
                        return a.rank_p2 < b.rank_p2;
                    }
                    return a.chars[1] < b.chars[1];
                }
                return a.chars[0] < b.chars[0];
            }
        }
        else
        {
            assert(b.index % 3 == 0);
            if (a.index % 3 == 1)
            {
                if (a.chars[0] == b.chars[0])
                    return a.rank_p1 < b.rank_p1;
                return a.chars[0] < b.chars[0];
            }
            else
            {
                if (a.chars[0] == b.chars[0])
                {
                    if (a.chars[1] == b.chars[1])
                    {
                        return a.rank_p2 < b.rank_p2;
                    }
                    return a.chars[1] < b.chars[1];
                }
                return a.chars[0] < b.chars[0];
            }
        }
    }
};

#include "prefix_doubling.hpp"

#define TIMER_START_PREPARE_FINAL_MERGE_STAGE(stage) mperf_measure.start_prepare_final_merge_stage(stage)
#define TIMER_STOP_PREPARE_FINAL_MERGE_STAGE(stage) mperf_measure.stop_prepare_final_merge_stage(stage)

class SuffixSorter
{
    static const int BLOCK_SIZE = 1024;
    static const size_t MAX_GRID_SIZE = 2048;

    using MemoryManager = SuffixArrayMemoryManager<NUM_GPUS, sa_index_t>;
    using MainStages = perf_rec::MainStages;
    using FinalMergeStages = perf_rec::PrepareFinalMergeStages;
    using Context = MultiGPUContext<NUM_GPUS>;

    struct SaGPU
    {
        size_t num_elements, offset;
        size_t pd_elements, pd_offset;
        PDArrays pd_ptr;
        PrepareS12Arrays prepare_S12_ptr;
        PrepareS0Arrays prepare_S0_ptr;
        MergeS12S0Arrays merge_ptr;
    };

    Context& mcontext;
    MemoryManager mmemory_manager;
    MultiSplit<NUM_GPUS> mmulti_split;
    All2All<NUM_GPUS> mall2all;
    std::array<SaGPU, NUM_GPUS> mgpus;

    SuffixArrayPerformanceMeasurements mperf_measure;

    PrefixDoublingSuffixSorter mpd_sorter;

    char* minput;
    size_t minput_len, mreserved_len, mpd_reserved_len, ms0_reserved_len, mper_gpu, mpd_per_gpu;
    size_t mpd_per_gpu_max_bit;
    size_t mtook_pd_iterations;

public:
    SuffixSorter(Context& context, size_t len, char* input)
        : mcontext(context), mmemory_manager(context),
        mmulti_split(context), mall2all(context),
        mperf_measure(32),
        mpd_sorter(mcontext, mmemory_manager, mmulti_split, mall2all, mperf_measure),
        minput(input), minput_len(len)
    {
    }

    void do_sa()
    {

        // TIMER_START_MAIN_STAGE(MainStages::Copy_Input);
        copy_input();
        //
        // mcontext.sync_all_streams();
        // printf("[%lu] Copy Input\n", world_rank());
        // comm_world().barrier();
        //

        TIMERSTART(Total);
        // TIMER_STOP_MAIN_STAGE(MainStages::Copy_Input);

        TIMER_START_MAIN_STAGE(MainStages::Produce_KMers);
        produce_kmers();
        //
        // mcontext.sync_all_streams();
        // printf("[%lu] Produce kmers\n", world_rank());
        // comm_world().barrier();
        //

        TIMER_STOP_MAIN_STAGE(MainStages::Produce_KMers);

        //            mpd_sorter.dump("After K-Mers");

        mtook_pd_iterations = mpd_sorter.sort(4);
        // printf("[%lu] sort done\n", world_rank());
        // comm_world().barrier();
        // auto& t = kamping::measurements::timer();
        // t.aggregate_and_print(
        //     kamping::measurements::SimpleJsonPrinter{ std::cout, {} });
        // std::cout << std::endl;
        // t.aggregate_and_print(kamping::measurements::FlatPrinter{});
        // std::cout << std::endl;
        //            mpd_sorter.dump("done");
        TIMER_START_MAIN_STAGE(MainStages::Prepare_S12_for_Merge);
        prepare_S12_for_merge();
        //
        // mcontext.sync_all_streams();
        // printf("[%lu] prepare s12 for merge done\n", world_rank());
        // comm_world().barrier();
        //

        TIMER_STOP_MAIN_STAGE(MainStages::Prepare_S12_for_Merge);
        TIMER_START_MAIN_STAGE(MainStages::Prepare_S0_for_Merge);
        prepare_S0_for_merge();
        //
        // mcontext.sync_all_streams();
        // printf("[%lu] prepare s0 for merge done\n", world_rank());
        // comm_world().barrier();
        //
        TIMER_STOP_MAIN_STAGE(MainStages::Prepare_S0_for_Merge);
        TIMER_START_MAIN_STAGE(MainStages::Final_Merge);
        final_merge();
        //
        // mcontext.sync_all_streams();
        // printf("[%lu] final merge done\n", world_rank());
        // comm_world().barrier();
        //
        TIMER_STOP_MAIN_STAGE(MainStages::Final_Merge);
        // TIMER_START_MAIN_STAGE(MainStages::Copy_Results);
        TIMERSTOP(Total);
        mperf_measure.done();

        copy_result_to_host();
        //
        // mcontext.sync_all_streams();
        // printf("[%lu] complete\n", world_rank());
        // comm_world().barrier();
        //
        // TIMER_STOP_MAIN_STAGE(MainStages::Copy_Results);
    }

    const sa_index_t* get_result() const
    {
        return mmemory_manager.get_h_result();
    }

    SuffixArrayPerformanceMeasurements& get_perf_measurements()
    {
        return mperf_measure;
    }

    void done()
    {
        mmemory_manager.free();
    }

    void alloc()
    {
        // mper_gpu how much data for one gpu
        mper_gpu = SDIV(minput_len, NUM_GPUS);
        ASSERT_MSG(mper_gpu >= 3, "Please give me more input.");

        // Ensure each gpu has a multiple of 3 because of triplets.
        mper_gpu = SDIV(mper_gpu, 3) * 3;
        printf("minput_len: %lu, mper_gpu %lu\n", minput_len, mper_gpu);
        ASSERT(minput_len > (NUM_GPUS - 1) * mper_gpu + 3); // Because of merge
        size_t last_gpu_elems = minput_len - (NUM_GPUS - 1) * mper_gpu;
        ASSERT(last_gpu_elems <= mper_gpu); // Because of merge.

        mreserved_len = SDIV(std::max(last_gpu_elems, mper_gpu) + 8, 12) * 12; // Ensure there are 12 elems more space.
        mreserved_len = std::max(mreserved_len, 1024ul);                       // Min len because of temp memory for CUB.

        mpd_reserved_len = SDIV(mreserved_len, 3) * 2;

        ms0_reserved_len = mreserved_len - mpd_reserved_len;

        auto cub_temp_mem = get_needed_cub_temp_memory(ms0_reserved_len, mpd_reserved_len);

        // Can do it this way since CUB temp memory is limited for large inputs.
        ms0_reserved_len = std::max(ms0_reserved_len, SDIV(cub_temp_mem.first, sizeof(MergeStageSuffix)));
        mpd_reserved_len = std::max(mpd_reserved_len, SDIV(cub_temp_mem.second, sizeof(MergeStageSuffix)));

        mmemory_manager.alloc(minput_len, mreserved_len, mpd_reserved_len, ms0_reserved_len, true);

        mpd_per_gpu = mper_gpu / 3 * 2;
        mpd_per_gpu_max_bit = std::min(sa_index_t(log2(float(mpd_per_gpu))) + 1, sa_index_t(sizeof(sa_index_t) * 8));

        size_t pd_total_len = 0, offset = 0, pd_offset = 0;
        for (uint i = 0; i < NUM_GPUS - 1; i++)
        {
            mgpus[i].num_elements = mper_gpu;
            mgpus[i].pd_elements = mpd_per_gpu;
            mgpus[i].offset = offset;
            mgpus[i].pd_offset = pd_offset;
            pd_total_len += mgpus[i].pd_elements;
            init_gpu_ptrs(i);
            offset += mper_gpu;
            pd_offset += mpd_per_gpu;
        }

        mgpus.back().num_elements = last_gpu_elems;
        // FIXME: Isn't this just...: last_gpu_elems / 3 * 2 + ((last_gpu_elems % 3) == 2);
        mgpus.back().pd_elements = last_gpu_elems / 3 * 2 + (((last_gpu_elems % 3) != 0) ? ((last_gpu_elems - 1) % 3) : 0);
        mgpus.back().offset = offset;
        mgpus.back().pd_offset = pd_offset;

        // Because of fixup.
        ASSERT(mgpus.back().pd_elements >= 4);

        pd_total_len += mgpus.back().pd_elements;
        init_gpu_ptrs(NUM_GPUS - 1);

        printf("Every node gets %zu (%zu) elements, last node: %zu (%zu), reserved len: %zu.\n", mper_gpu,
            mpd_per_gpu, last_gpu_elems, mgpus.back().pd_elements, mreserved_len);

        mpd_sorter.init(pd_total_len, mpd_per_gpu, mgpus.back().pd_elements, mpd_reserved_len);
    }

    void print_pd_stats() const
    {
        mpd_sorter.print_stats(mtook_pd_iterations);
    }

private:
    void init_gpu_ptrs(uint i)
    {
        mgpus[i].pd_ptr = mmemory_manager.get_pd_arrays(i);
        mgpus[i].prepare_S12_ptr = mmemory_manager.get_prepare_S12_arrays(i);
        mgpus[i].prepare_S0_ptr = mmemory_manager.get_prepare_S0_arrays(i);
        mgpus[i].merge_ptr = mmemory_manager.get_merge_S12_S0_arrays(i);
    }

    std::pair<size_t, size_t> get_needed_cub_temp_memory(size_t S0_count, size_t S12_count) const
    {
        hipcub::DoubleBuffer<uint64_t> keys(nullptr, nullptr);
        hipcub::DoubleBuffer<uint64_t> values(nullptr, nullptr);

        size_t temp_storage_size_S0 = 0;
        size_t temp_storage_size_S12 = 0;
        hipError_t err = hipcub::DeviceRadixSort::SortPairs(nullptr, temp_storage_size_S0,
            keys, values, S0_count, 0, 40);
        CUERR_CHECK(err);
        err = hipcub::DeviceRadixSort::SortPairs(nullptr, temp_storage_size_S12,
            keys, values, S12_count, 0, 40);
        CUERR_CHECK(err);

        return { temp_storage_size_S0, temp_storage_size_S12 };
    }

    void copy_input()
    {
        using kmer_t = uint64_t;
        // for (uint gpu_index = 0; gpu_index < NUM_GPUS; ++gpu_index)
        //{
        auto gpu_index = world_rank();
        SaGPU& gpu = mgpus[gpu_index];

        // Need the halo to the right for kmers...
        size_t copy_len = std::min(gpu.num_elements + sizeof(kmer_t), minput_len - gpu.offset);

        //(mcontext.get_device_id(gpu_index));
        hipMemcpyAsync(gpu.pd_ptr.Input, minput, copy_len, hipMemcpyHostToDevice,
            mcontext.get_gpu_default_stream(gpu_index));
        CUERR;
        if (gpu_index == NUM_GPUS - 1)
        {
            hipMemsetAsync(gpu.pd_ptr.Input + gpu.num_elements, 0, sizeof(kmer_t),
                mcontext.get_gpu_default_stream(gpu_index));
            CUERR;
        }
        //}

        mcontext.sync_default_streams();
    }

    void produce_kmers()
    {
        // for (uint gpu_index = 0; gpu_index < NUM_GPUS; ++gpu_index)
        //{
        auto gpu_index = world_rank();
        SaGPU& gpu = mgpus[gpu_index];

        //(mcontext.get_device_id(gpu_index));
        //                kernels::produce_index_kmer_tuples _KLC_SIMPLE_(gpu.num_elements, mcontext.get_gpu_default_stream(gpu_index))
        //                        ((char*)gpu.input, offset, gpu.pd_index, gpu.pd_kmers, gpu.num_elements); CUERR;
        kernels::produce_index_kmer_tuples_12_64 _KLC_SIMPLE_(gpu.num_elements, mcontext.get_gpu_default_stream(gpu_index))((char*)gpu.pd_ptr.Input, gpu.pd_offset, gpu.pd_ptr.Isa, reinterpret_cast<ulong1*>(gpu.pd_ptr.Sa_rank),
            SDIV(gpu.num_elements, 12) * 12);
        CUERR;
        //}
        if (gpu_index == NUM_GPUS - 1)
        {
            kernels::fixup_last_four_12_kmers_64 << <1, 4, 0, mcontext.get_gpu_default_stream(gpu_index) >> > (reinterpret_cast<ulong1*>(mgpus.back().pd_ptr.Sa_rank) + mgpus.back().pd_elements - 4);
        }
        mcontext.sync_default_streams();
    }

    void prepare_S12_for_merge()
    {
        std::array<MultiSplitNodeInfoT<sa_index_t, sa_index_t, sa_index_t>, NUM_GPUS> multi_split_node_info;
        std::array<All2AllNodeInfoT<MergeStageSuffixS12HalfKey, MergeStageSuffixS12HalfValue, sa_index_t>, NUM_GPUS> all2all_node_info;
        split_table_tt<sa_index_t, NUM_GPUS> split_table;
        std::array<sa_index_t, NUM_GPUS> dest_lens, src_lens;

        TIMER_START_PREPARE_FINAL_MERGE_STAGE(FinalMergeStages::S12_Multisplit);
        for (uint gpu_index = 0; gpu_index < NUM_GPUS; ++gpu_index)
        {
            SaGPU& gpu = mgpus[gpu_index];
            if (world_rank() == gpu_index)
            {

                // //(0);
                kernels::write_indices _KLC_SIMPLE_(gpu.pd_elements, mcontext.get_gpu_default_stream(gpu_index))((sa_index_t*)gpu.prepare_S12_ptr.S12_result, gpu.pd_elements);
                CUERR;
                mcontext.get_device_temp_allocator(gpu_index).init(gpu.prepare_S12_ptr.S12_buffer1,
                    mpd_reserved_len * sizeof(MergeStageSuffixS12));
            }

            multi_split_node_info[gpu_index].src_keys = gpu.prepare_S12_ptr.Isa;
            multi_split_node_info[gpu_index].src_values = (sa_index_t*)gpu.prepare_S12_ptr.S12_result;
            multi_split_node_info[gpu_index].src_len = gpu.pd_elements;

            multi_split_node_info[gpu_index].dest_keys = (sa_index_t*)gpu.prepare_S12_ptr.S12_buffer2;
            multi_split_node_info[gpu_index].dest_values = (sa_index_t*)gpu.prepare_S12_ptr.S12_result_half;
            multi_split_node_info[gpu_index].dest_len = gpu.pd_elements;
        }
        // printf("[%lu] mpd_per_gpu: %lu\n", world_rank(), mpd_per_gpu);
        S12PartitioningFunctor f(mpd_per_gpu, NUM_GPUS - 1);



        //
        mcontext.sync_default_streams();
        // comm_world().barrier();
        //
        // printf("[%lu] after write indices s12\n", world_rank());
        mmulti_split.execKVAsync(multi_split_node_info, split_table, src_lens, dest_lens, f);

        mcontext.sync_default_streams();
        // for (size_t src = 0; src < NUM_GPUS; src++)
        // {
        //     for (size_t dst = 0; dst < NUM_GPUS; dst++)
        //     {
        //         printf("[%lu] split_table[%lu][%lu]: %u\n", world_rank(), src, dst, split_table[src][dst]);
        //     }
        // }

        comm_world().barrier();
        // printf("[%lu] after execKVAsync s12\n", world_rank());

        TIMER_STOP_PREPARE_FINAL_MERGE_STAGE(FinalMergeStages::S12_Multisplit);

        TIMER_START_PREPARE_FINAL_MERGE_STAGE(FinalMergeStages::S12_Write_Out);

        // for (uint gpu_index = 0; gpu_index < NUM_GPUS; ++gpu_index)
        {
            uint gpu_index = world_rank();
            SaGPU& gpu = mgpus[gpu_index];
            //(mcontext.get_device_id(gpu_index));

            const sa_index_t* next_Isa = nullptr;      //= (gpu_index + 1 < NUM_GPUS) ? mgpus[gpu_index + 1].prepare_S12_ptr.Isa : nullptr;
            const unsigned char* next_Input = nullptr; //= (gpu_index + 1 < NUM_GPUS) ? mgpus[gpu_index + 1].prepare_S12_ptr.Input : nullptr;

            ncclGroupStart();
            if (gpu_index > 0)
            {
                std::span<sa_index_t> sbIsa(gpu.prepare_S12_ptr.Isa, 1);
                ncclSend(gpu.prepare_S12_ptr.Isa, 1, ncclUint32, gpu_index - 1, mcontext.get_nccl(), mcontext.get_streams(gpu_index)[gpu_index - 1]);
                // comm_world().isend(send_buf(sbIsa), send_count(1), tag(0), destination((size_t)gpu_index - 1));
                ncclSend(gpu.prepare_S12_ptr.Input, 1, ncclChar, gpu_index - 1, mcontext.get_nccl(), mcontext.get_streams(gpu_index)[gpu_index - 1]);

                std::span<const unsigned char> sbInput(gpu.prepare_S12_ptr.Input, 1);
                // comm_world().isend(send_buf(sbInput), send_count(1), tag(1), destination((size_t)gpu_index - 1));
            }
            if (gpu_index + 1 < NUM_GPUS)
            {
                sa_index_t* next_Isa = mcontext.get_device_temp_allocator(gpu_index).get<sa_index_t>(1);
                // std::span<sa_index_t> rbIsa(tempIsa, 1);
                ncclRecv(next_Isa, 1, ncclUint32, gpu_index + 1, mcontext.get_nccl(), mcontext.get_gpu_default_stream(gpu_index));

                // comm_world().recv(recv_buf(rbIsa), tag(0), recv_count(1));
                // next_Isa = tempIsa;
                unsigned char* next_Input = mcontext.get_device_temp_allocator(gpu_index).get<unsigned char>(1);
                // std::span<unsigned char> rbInput(tempInput, 1);
                ncclRecv(next_Input, 1, ncclChar, gpu_index + 1, mcontext.get_nccl(), mcontext.get_gpu_default_stream(gpu_index));
                // comm_world().recv(recv_buf(rbInput), tag(1), recv_count(1));
                // next_Input = tempInput;
            }
            ncclGroupEnd();
            kernels::prepare_S12_ind_kv _KLC_SIMPLE_(gpu.pd_elements, mcontext.get_gpu_default_stream(gpu_index))((sa_index_t*)gpu.prepare_S12_ptr.S12_result_half,
                gpu.prepare_S12_ptr.Isa, gpu.prepare_S12_ptr.Input,
                next_Isa, next_Input, gpu.offset, gpu.num_elements,
                mpd_per_gpu,
                gpu.prepare_S12_ptr.S12_buffer1, gpu.prepare_S12_ptr.S12_buffer1_half, gpu.pd_elements);
            CUERR;
        }

        for (uint gpu_index = 0; gpu_index < NUM_GPUS; ++gpu_index)
        {
            SaGPU& gpu = mgpus[gpu_index];
            //                printf("GPU %u, sr    c: %u, dest: %u.\n", gpu_index, src_lens[gpu_index], dest_lens[gpu_index]);
            all2all_node_info[gpu_index].src_keys = gpu.prepare_S12_ptr.S12_buffer1;
            all2all_node_info[gpu_index].src_values = gpu.prepare_S12_ptr.S12_buffer1_half;
            all2all_node_info[gpu_index].src_len = gpu.pd_elements;

            all2all_node_info[gpu_index].dest_keys = gpu.prepare_S12_ptr.S12_buffer2;
            all2all_node_info[gpu_index].dest_values = gpu.prepare_S12_ptr.S12_buffer2_half;
            all2all_node_info[gpu_index].dest_len = gpu.pd_elements;

            all2all_node_info[gpu_index].temp_keys = reinterpret_cast<MergeStageSuffixS12HalfKey*>(gpu.prepare_S12_ptr.S12_result);
            all2all_node_info[gpu_index].temp_values = gpu.prepare_S12_ptr.S12_result_half;
            all2all_node_info[gpu_index].temp_len = mpd_reserved_len; // not sure...
        }
        mcontext.sync_default_streams();
        //
        // mcontext.get_device_temp_allocator(world_rank()).reset();
        //
        TIMER_STOP_PREPARE_FINAL_MERGE_STAGE(FinalMergeStages::S12_Write_Out);
        TIMER_START_PREPARE_FINAL_MERGE_STAGE(FinalMergeStages::S12_All2All);

        //            dump_prepare_s12("After split");
        comm_world().barrier();
        // printf("[%lu] after prepare_S12_ind_kv s12\n", world_rank());
        mall2all.execKVAsync(all2all_node_info, split_table, true);
        mcontext.sync_all_streams_mpi_safe();
        comm_world().barrier();
        TIMER_STOP_PREPARE_FINAL_MERGE_STAGE(FinalMergeStages::S12_All2All);
        // printf("[%lu] all2all s12\n", world_rank());

        //            dump_prepare_s12("After all2all");

        TIMER_START_PREPARE_FINAL_MERGE_STAGE(FinalMergeStages::S12_Write_Into_Place);

        // for (uint gpu_index = 0; gpu_index < NUM_GPUS; ++gpu_index)
        {
            uint gpu_index = world_rank();
            const uint SORT_DOWN_TO_BIT = 11;

            SaGPU& gpu = mgpus[gpu_index];
            //(mcontext.get_device_id(gpu_index));

            hipcub::DoubleBuffer<uint64_t> keys(reinterpret_cast<uint64_t*>(gpu.prepare_S12_ptr.S12_buffer2),
                reinterpret_cast<uint64_t*>(gpu.prepare_S12_ptr.S12_buffer1));
            hipcub::DoubleBuffer<uint64_t> values(reinterpret_cast<uint64_t*>(gpu.prepare_S12_ptr.S12_buffer2_half),
                reinterpret_cast<uint64_t*>(gpu.prepare_S12_ptr.S12_buffer1_half));
            if (SORT_DOWN_TO_BIT < mpd_per_gpu_max_bit)
            {
                size_t temp_storage_size = 0;
                hipError_t err = hipcub::DeviceRadixSort::SortPairs(nullptr, temp_storage_size, keys, values, gpu.pd_elements,
                    SORT_DOWN_TO_BIT, mpd_per_gpu_max_bit);
                CUERR_CHECK(err);
                //                printf("Needed temp storage: %zu, provided %zu.\n", temp_storage_size, ms0_reserved_len*sizeof(MergeStageSuffix));
                ASSERT(temp_storage_size <= mpd_reserved_len * sizeof(MergeStageSuffix));
                err = hipcub::DeviceRadixSort::SortPairs(gpu.prepare_S12_ptr.S12_result, temp_storage_size,
                    keys, values, gpu.pd_elements, SORT_DOWN_TO_BIT, mpd_per_gpu_max_bit,
                    mcontext.get_gpu_default_stream(gpu_index));
                CUERR_CHECK(err);
            }

            // printf("[%lu] S12_Write_Into_Place\n", world_rank());
            // mcontext.sync_default_stream_mpi_safe();
            // comm_world().barrier();
            //                kernels::combine_S12_kv_non_coalesced _KLC_SIMPLE_(gpu.pd_elements, mcontext.get_gpu_default_stream(gpu_index))
            //                        (reinterpret_cast<MergeStageSuffixS12HalfKey*> (gpu.prepare_S12_ptr.S12_buffer2),
            //                         reinterpret_cast<MergeStageSuffixS12HalfValue*> ( gpu.prepare_S12_ptr.S12_buffer2_half),
            //                         gpu.prepare_S12_ptr.S12_result, gpu.pd_elements); CUERR

            kernels::combine_S12_kv_shared<BLOCK_SIZE, 2> _KLC_SIMPLE_ITEMS_PER_THREAD_(gpu.pd_elements, 2, mcontext.get_gpu_default_stream(gpu_index))(reinterpret_cast<MergeStageSuffixS12HalfKey*>(keys.Current()),
                reinterpret_cast<MergeStageSuffixS12HalfValue*>(values.Current()),
                gpu.prepare_S12_ptr.S12_result, gpu.pd_elements);
            CUERR;
        }
        mcontext.sync_default_stream_mpi_safe();
        comm_world().barrier();
        // mcontext.sync_default_streams();

        TIMER_STOP_PREPARE_FINAL_MERGE_STAGE(FinalMergeStages::S12_Write_Into_Place);

        //            dump_prepare_s12("After preparing S12");
        //            dump_final_merge("After preparing S12");
    }

    void prepare_S0_for_merge()
    {
        using merge_types = crossGPUReMerge::mergeTypes<MergeStageSuffixS0HalfKey, MergeStageSuffixS0HalfValue>;
        using MergeManager = crossGPUReMerge::ReMergeManager<NUM_GPUS, merge_types, ReMergeTopology>;
        using MergeNodeInfo = crossGPUReMerge::MergeNodeInfo<merge_types>;

        auto host_temp_mem = mmemory_manager.get_host_temp_mem();

        QDAllocator host_pinned_allocator(host_temp_mem.first, host_temp_mem.second);

        std::array<MergeNodeInfo, NUM_GPUS> merge_nodes_info;

        std::array<bool, NUM_GPUS> is_buffer_2_current = { false };

        TIMER_START_PREPARE_FINAL_MERGE_STAGE(FinalMergeStages::S0_Write_Out_And_Sort);

        // for (uint gpu_index = 0; gpu_index < NUM_GPUS; ++gpu_index)
        {
            uint gpu_index = world_rank();
            SaGPU& gpu = mgpus[gpu_index];
            // //(mcontext.get_device_id(gpu_index));
            size_t count = gpu.num_elements - gpu.pd_elements;

            kernels::prepare_S0 _KLC_SIMPLE_(count, mcontext.get_gpu_default_stream(gpu_index))(gpu.prepare_S0_ptr.Isa, gpu.prepare_S0_ptr.Input, gpu.offset,
                gpu.num_elements, gpu.pd_elements,
                gpu_index == NUM_GPUS - 1,
                reinterpret_cast<MergeStageSuffixS0HalfKey*>(gpu.prepare_S0_ptr.S0_buffer1_keys),
                gpu.prepare_S0_ptr.S0_buffer1_values,
                count);
            CUERR;
            hipcub::DoubleBuffer<uint64_t> keys(reinterpret_cast<uint64_t*>(gpu.prepare_S0_ptr.S0_buffer1_keys),
                reinterpret_cast<uint64_t*>(gpu.prepare_S0_ptr.S0_buffer2_keys));
            hipcub::DoubleBuffer<uint64_t> values(reinterpret_cast<uint64_t*>(gpu.prepare_S0_ptr.S0_buffer1_values),
                reinterpret_cast<uint64_t*>(gpu.prepare_S0_ptr.S0_buffer2_values));

            size_t temp_storage_size = 0;
            hipError_t err = hipcub::DeviceRadixSort::SortPairs(nullptr, temp_storage_size, keys, values, count, 0, 40);
            CUERR_CHECK(err);
            //                printf("Needed temp storage: %zu, provided %zu.\n", temp_storage_size, ms0_reserved_len*sizeof(MergeStageSuffix));
            ASSERT(temp_storage_size <= ms0_reserved_len * sizeof(MergeStageSuffix));
            err = hipcub::DeviceRadixSort::SortPairs(gpu.prepare_S0_ptr.S0_result, temp_storage_size,
                keys, values, count, 0, 40, mcontext.get_gpu_default_stream(gpu_index));
            CUERR_CHECK(err);

            is_buffer_2_current[gpu_index] = keys.Current() == reinterpret_cast<uint64_t*>(gpu.prepare_S0_ptr.S0_buffer2_keys);

            // merge_nodes_info[gpu_index] = { count, ms0_reserved_len, gpu_index,
            //                                is_buffer_2_current[gpu_index] ? gpu.prepare_S0_ptr.S0_buffer2_keys
            //                                                               : gpu.prepare_S0_ptr.S0_buffer1_keys,
            //                                is_buffer_2_current[gpu_index] ? gpu.prepare_S0_ptr.S0_buffer2_values
            //                                                               : gpu.prepare_S0_ptr.S0_buffer1_values,
            //                                is_buffer_2_current[gpu_index] ? gpu.prepare_S0_ptr.S0_buffer1_keys
            //                                                               : gpu.prepare_S0_ptr.S0_buffer2_keys,
            //                                is_buffer_2_current[gpu_index] ? gpu.prepare_S0_ptr.S0_buffer1_values
            //                                                               : gpu.prepare_S0_ptr.S0_buffer2_values,
            //                                reinterpret_cast<MergeStageSuffixS0HalfKey*>(gpu.prepare_S0_ptr.S0_result),
            //                                gpu.prepare_S0_ptr.S0_result_2nd_half };

            mcontext.get_device_temp_allocator(gpu_index).init(reinterpret_cast<MergeStageSuffixS0HalfKey*>(gpu.prepare_S0_ptr.S0_result),
                ms0_reserved_len * sizeof(MergeStageSuffixS0));
        }
        for (uint gpu_index = 0; gpu_index < NUM_GPUS; ++gpu_index)
        {
            SaGPU& gpu = mgpus[gpu_index];
            size_t count = gpu.num_elements - gpu.pd_elements;
            // send which current is used (only for in node merges)
            comm_world().bcast(send_recv_buf(std::span<bool>(&is_buffer_2_current[gpu_index], 1)), send_recv_count(1), root((size_t)gpu_index));
            merge_nodes_info[gpu_index] = { count, ms0_reserved_len, gpu_index,
                                           is_buffer_2_current[gpu_index] ? gpu.prepare_S0_ptr.S0_buffer2_keys
                                                                          : gpu.prepare_S0_ptr.S0_buffer1_keys,
                                           is_buffer_2_current[gpu_index] ? gpu.prepare_S0_ptr.S0_buffer2_values
                                                                          : gpu.prepare_S0_ptr.S0_buffer1_values,
                                           is_buffer_2_current[gpu_index] ? gpu.prepare_S0_ptr.S0_buffer1_keys
                                                                          : gpu.prepare_S0_ptr.S0_buffer2_keys,
                                           is_buffer_2_current[gpu_index] ? gpu.prepare_S0_ptr.S0_buffer1_values
                                                                          : gpu.prepare_S0_ptr.S0_buffer2_values,
                                           reinterpret_cast<MergeStageSuffixS0HalfKey*>(gpu.prepare_S0_ptr.S0_result),
                                           gpu.prepare_S0_ptr.S0_result_2nd_half };
        }

        //            dump_prepare_s0("Before S0 merge");

        MergeManager merge_manager(mcontext, host_pinned_allocator);

        merge_manager.set_node_info(merge_nodes_info);

        std::vector<crossGPUReMerge::MergeRange> ranges;
        ranges.push_back({ 0, 0, (sa_index_t)NUM_GPUS - 1, (sa_index_t)(mgpus.back().num_elements - mgpus.back().pd_elements) });

        mcontext.sync_default_stream_mpi_safe();
        // printf("[%lu] after S0_Write_Out_And_Sort s0\n", world_rank());
        TIMER_STOP_PREPARE_FINAL_MERGE_STAGE(FinalMergeStages::S0_Write_Out_And_Sort);

        TIMER_START_PREPARE_FINAL_MERGE_STAGE(FinalMergeStages::S0_Merge);
        merge_manager.merge(ranges, S0Comparator());

        mcontext.sync_all_streams_mpi_safe();
        comm_world().barrier();
        // printf("[%lu] after merge s0\n", world_rank());
        TIMER_STOP_PREPARE_FINAL_MERGE_STAGE(FinalMergeStages::S0_Merge);

        TIMER_START_PREPARE_FINAL_MERGE_STAGE(FinalMergeStages::S0_Combine);

        // for (uint gpu_index = 0; gpu_index < NUM_GPUS; ++gpu_index)
        {
            uint gpu_index = world_rank();
            SaGPU& gpu = mgpus[gpu_index];
            //(mcontext.get_device_id(gpu_index));

            size_t count = gpu.num_elements - gpu.pd_elements;

            const MergeStageSuffixS0HalfKey* sorted_and_merged_keys = is_buffer_2_current[gpu_index] ? gpu.prepare_S0_ptr.S0_buffer2_keys : gpu.prepare_S0_ptr.S0_buffer1_keys;

            const MergeStageSuffixS0HalfValue* sorted_and_merged_values = is_buffer_2_current[gpu_index] ? gpu.prepare_S0_ptr.S0_buffer2_values : gpu.prepare_S0_ptr.S0_buffer1_values;

            kernels::combine_S0_kv _KLC_SIMPLE_(count, mcontext.get_gpu_default_stream(gpu_index))(sorted_and_merged_keys, sorted_and_merged_values, gpu.prepare_S0_ptr.S0_result, count);
            CUERR;
        }
        mcontext.sync_default_stream_mpi_safe();
        // printf("[%lu] after s0\n", world_rank());
        TIMER_STOP_PREPARE_FINAL_MERGE_STAGE(FinalMergeStages::S0_Combine);
        //            dump_final_merge("before final merge");
    }

    void final_merge()
    {
        distrib_merge::DistributedArray<MergeStageSuffix, int, sa_index_t, NUM_GPUS> inp_S12, inp_S0, result;

        for (uint gpu_index = 0; gpu_index < NUM_GPUS; ++gpu_index)
        {

            SaGPU& gpu = mgpus[gpu_index];

            const size_t S0_count = gpu.num_elements - gpu.pd_elements;
            const size_t S12_count = gpu.pd_elements;
            const size_t result_count = gpu.num_elements;
            inp_S12[gpu_index] = { gpu_index, (sa_index_t)S12_count, gpu.merge_ptr.S12_result, nullptr, nullptr, nullptr };
            inp_S0[gpu_index] = { gpu_index, (sa_index_t)S0_count, gpu.merge_ptr.S0_result, nullptr, nullptr, nullptr };
            result[gpu_index] = { gpu_index, (sa_index_t)result_count, gpu.merge_ptr.S12_result, nullptr, gpu.merge_ptr.buffer, nullptr };
            if (world_rank() == gpu_index)
            {
                mcontext.get_device_temp_allocator(gpu_index).init(gpu.merge_ptr.remaining_storage,
                    gpu.merge_ptr.remaining_storage_size);
            }
        }
        // printf("[%lu] final merge\n", world_rank());
        auto h_temp_mem = mmemory_manager.get_host_temp_mem();
        QDAllocator qd_alloc_h_temp(h_temp_mem.first, h_temp_mem.second);
        distrib_merge::DistributedMerge<MergeStageSuffix, int, sa_index_t, NUM_GPUS, DistribMergeTopology>::
            merge_async(inp_S12, inp_S0, result, MergeCompFunctor(), false, mcontext, qd_alloc_h_temp);

        mcontext.sync_default_streams();
        // printf("[%lu] after merge_async\n", world_rank());

        // printf("[%lu] merge async done\n", world_rank());
        // comm_world().barrier();
        //            dump_final_merge("after final merge");

        // for (uint gpu_index = 0; gpu_index < NUM_GPUS; ++gpu_index)
        {
            uint gpu_index = world_rank();
            SaGPU& gpu = mgpus[gpu_index];
            // //(mcontext.get_device_id(gpu_index));
            kernels::from_merge_suffix_to_index _KLC_SIMPLE_(gpu.num_elements, mcontext.get_gpu_default_stream(gpu_index))(gpu.merge_ptr.S12_result, gpu.merge_ptr.result, gpu.num_elements);
            CUERR;
        }
        mcontext.sync_default_streams();
    }

    void copy_result_to_host()
    {
        sa_index_t* h_result = mmemory_manager.get_h_result();
        // for (uint gpu_index = 0; gpu_index < NUM_GPUS; ++gpu_index)
        //{
        uint gpu_index = world_rank();
        SaGPU& gpu = mgpus[gpu_index];
        //(mcontext.get_device_id(gpu_index));
        hipMemcpyAsync(h_result, gpu.merge_ptr.result, gpu.num_elements * sizeof(sa_index_t),
            hipMemcpyDeviceToHost, mcontext.get_gpu_default_stream(gpu_index));
        CUERR;
        mcontext.sync_gpu_default_stream(gpu_index);
        int ierr;
        MPI_File outputFile;
        ierr = MPI_File_open(MPI_COMM_WORLD, "outputData",
            MPI_MODE_CREATE | MPI_MODE_WRONLY,
            MPI_INFO_NULL, &outputFile);
        if (ierr != MPI_SUCCESS) {
            fprintf(stderr, "[%lu] Error opening file\n", world_rank());
            MPI_Abort(MPI_COMM_WORLD, ierr);
        }
        MPI_Offset offset = gpu.offset * sizeof(sa_index_t);
        ierr = MPI_File_write_at_all(outputFile, offset, h_result, gpu.num_elements, MPI_UINT32_T, MPI_STATUS_IGNORE);
        if (ierr != MPI_SUCCESS) {
            fprintf(stderr, "[%lu] Error in MPI_File_write_at_all\n", world_rank());
            MPI_Abort(MPI_COMM_WORLD, ierr);
        }
        MPI_File_close(&outputFile);

        // MPI_File outputFile;
        // MPI_File_open(MPI_COMM_WORLD, "outputData",
        //     MPI_MODE_CREATE | MPI_MODE_WRONLY,
        //     MPI_INFO_NULL, &outputFile);
        // MPI_File_write_at_all(outputFile, gpu.offset, h_result, gpu.num_elements, MPI_UINT32_T, MPI_STATUS_IGNORE);
        // // MPI_File_write_at(outputFile, gpu.offset, h_result, gpu.num_elements, MPI_UINT32_T, MPI_STATUS_IGNORE);

        // MPI_File_close(&outputFile);

        //}
        // mcontext.sync_default_streams();

        // std::vector<sa_index_t> recv;
        // recv.clear();
        // std::span<sa_index_t> sb(h_result + gpu.offset, gpu.num_elements);
        // auto [sendCounts] = comm_world().gatherv(send_buf(sb), recv_buf<resize_to_fit>(recv), recv_counts_out());
        // int sumCounts = 0;
        // int i = 0;
        // for (auto count : sendCounts) {
        //     ASSERT(count == mgpus[i].num_elements);
        //     memcpy(h_result + mgpus[i].offset, recv.data() + sumCounts, sizeof(sa_index_t) * count);
        //     sumCounts += count;
        //     i++;
        // }
        // for (int gpu_index = 0; gpu_index < NUM_GPUS; ++gpu_index) {
        //     std::span<sa_index_t> buffer(h_result + gpu.offset, gpu.num_elements);
        //     comm_world().bcast(send_recv_buf(buffer), root(gpu_index));
        // }
        // std::span<sa_index_t> rb(h_result, );
    }

#ifdef ENABLE_DUMPING
    static inline void print_merge12(sa_index_t index, const MergeStageSuffixS12HalfKey& s12k,
        const MergeStageSuffixS12HalfValue& s12v)
    {
        printf("%7u. Index: %7u, own rank: %7u, rank +1/+2: %7u, c: %2x (%c), c[i+1]: %2x (%c)\n",
            index, s12k.index, s12k.own_rank, s12v.rank_p1p2, s12v.chars[0], s12v.chars[0],
            s12v.chars[1], s12v.chars[1]);
    }

    static inline void print_merge0_half(sa_index_t index, const MergeStageSuffixS0HalfKey& s0k,
        const MergeStageSuffixS0HalfValue& s0v)
    {
        printf("%7u. Index: %7u, first char: %2x (%c), c[i+1]: %2x (%c), rank[i+1]: %7u, rank[i+2]: %7u\n",
            index, s0v.index, s0k.chars[0], s0k.chars[0], s0k.chars[1], s0k.chars[1],
            s0k.rank_p1, s0v.rank_p2);
    }

    static inline void print_final_merge_suffix(sa_index_t index, const MergeStageSuffix& suff)
    {
        printf("%7u. Index: %7u, first char: %2x (%c), c[i+1]: %2x (%c), rank[i+1]: %7u, rank[i+2]: %7u\n",
            index, suff.index, suff.chars[0], suff.chars[0], suff.chars[1], suff.chars[1],
            suff.rank_p1, suff.rank_p2);
    }

    void dump_prepare_s12(const char* caption = nullptr)
    {
        if (caption)
        {
            printf("\n%s:\n", caption);
        }
        for (uint g = 0; g < NUM_GPUS; ++g)
        {
            mmemory_manager.copy_down_for_inspection(g);
            printf("\nGPU %u:\nBuffer1:\n", g);
            size_t limit = mgpus[g].pd_elements;
            const PrepareS12Arrays& arr = mmemory_manager.get_host_prepare_S12_arrays();
            for (int i = 0; i < limit; ++i)
            {
                print_merge12(i, arr.S12_buffer1[i], arr.S12_buffer1_half[i]);
            }
            printf("Buffer2:\n");
            for (int i = 0; i < limit; ++i)
            {
                print_merge12(i, arr.S12_buffer2[i], arr.S12_buffer2_half[i]);
            }
            printf("Result-buffer:\n");
            for (int i = 0; i < limit; ++i)
            {
                print_final_merge_suffix(i, arr.S12_result[i]);
            }
        }
    }

    void dump_prepare_s0(const char* caption = nullptr)
    {
        if (caption)
        {
            printf("\n%s:\n", caption);
        }
        for (uint g = 0; g < NUM_GPUS; ++g)
        {
            mmemory_manager.copy_down_for_inspection(g);
            printf("\nGPU %u:\nBuffer1:\n", g);
            size_t limit = mgpus[g].num_elements - mgpus[g].pd_elements;
            const PrepareS0Arrays& arr = mmemory_manager.get_host_prepare_S0_arrays();
            for (int i = 0; i < limit; ++i)
            {
                print_merge0_half(i, arr.S0_buffer1_keys[i], arr.S0_buffer1_values[i]);
            }
            printf("Buffer2:\n");
            for (int i = 0; i < limit; ++i)
            {
                print_merge0_half(i, reinterpret_cast<const MergeStageSuffixS0HalfKey*>(arr.S0_buffer2_keys)[i],
                    arr.S0_buffer2_values[i]);
            }
            printf("Result-buffer:\n");
            for (int i = 0; i < limit; ++i)
            {
                print_final_merge_suffix(i, arr.S0_result[i]);
            }
        }
    }

    void dump_final_merge(const char* caption = nullptr)
    {
        if (caption)
        {
            printf("\n%s:\n", caption);
        }
        for (uint g = 0; g < NUM_GPUS; ++g)
        {
            SaGPU& gpu = mgpus[g];

            mmemory_manager.copy_down_for_inspection(g);

            printf("\nGPU %u:\nS12_result:\n", g);
            const MergeS12S0Arrays& arr = mmemory_manager.get_host_merge_S12_S0_arrays();

            for (int i = 0; i < gpu.pd_elements; ++i)
            {
                if (i == 10 && gpu.pd_elements > 20)
                    i = gpu.pd_elements - 10;
                print_final_merge_suffix(i, arr.S12_result[i]);
            }
            printf("S0_result:\n");
            for (int i = 0; i < gpu.num_elements - gpu.pd_elements; ++i)
            {
                if (i == 10 && (gpu.num_elements - gpu.pd_elements) > 20)
                    i = (gpu.num_elements - gpu.pd_elements) - 10;
                print_final_merge_suffix(i, arr.S0_result[i]);
            }
            //                printf("Buffer:\n");
            //                for (int i = 0; i < gpu.num_elements; ++i) {
            //                    if (i == 10 && gpu.num_elements > 20)
            //                        i = gpu.num_elements-10;
            //                    print_final_merge_suffix(i, arr.buffer[i]);
            //                }
        }
    }
#endif
};

void print_device_info()
{
    int nDevices;

    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; ++i)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
            prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
            prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n",
            2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
        printf("  Major.minor: %d.%d\n",
            prop.major, prop.minor);
        printf("  Max grid size: %d, %d, %d\n",
            prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
        printf("  Max threads dim (per block): %d, %d, %d\n",
            prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("  Max thread per block: %d\n",
            prop.maxThreadsPerBlock);
        printf("  Warp size: %d\n",
            prop.warpSize);
        printf("  Global mem: %zd kB\n",
            prop.totalGlobalMem / 1024);
        printf("  Const mem: %zd kB\n",
            prop.totalConstMem / 1024);
        printf("  Asynchronous engines: %d\n",
            prop.asyncEngineCount);
        printf("  Unified addressing: %d\n",
            prop.unifiedAddressing);
    }
}

void ncclMeasure(MultiGPUContext<NUM_GPUS>& context)
{
    using namespace kamping;
    std::random_device rd;
    std::mt19937 g(rd());
    std::uniform_int_distribution<std::mt19937::result_type> randomDist(0, UINT32_MAX);
    const int rounds = 26;
    std::array<double, rounds> alg_bandwidth;
    ncclComm_t nccl_comm = context.get_nccl();
    const int start_offset = 0;
    for (int i = 0; i < rounds; i++)
    {

        size_t per_gpu = NUM_GPUS << i;
        sa_index_t* A = (sa_index_t*)malloc(per_gpu * sizeof(sa_index_t));
        for (size_t j = 0; j < per_gpu; j++)
        {
            A[j] = randomDist(g);
        }
        sa_index_t* d_A_send;
        sa_index_t* d_A_recv;
        sa_index_t send_size = per_gpu / NUM_GPUS;
        hipMalloc(&d_A_send, sizeof(sa_index_t) * per_gpu);
        hipMalloc(&d_A_recv, sizeof(sa_index_t) * per_gpu);
        hipMemset(d_A_recv, 0, sizeof(sa_index_t) * per_gpu);
        hipMemcpy(d_A_send, A, sizeof(sa_index_t) * per_gpu, hipMemcpyHostToDevice);

        // warm up
        for (int loop = 0; loop < 1; loop++)
        {
            ncclGroupStart();
            for (size_t src_gpu = 0; src_gpu < NUM_GPUS; src_gpu++)
            {
                for (size_t dst_gpu = 0; dst_gpu < NUM_GPUS; dst_gpu++)
                {
                    if (src_gpu == world_rank())
                    {
                        ncclSend(d_A_send + dst_gpu * send_size, sizeof(sa_index_t) * send_size, ncclChar, dst_gpu, nccl_comm, context.get_streams(src_gpu)[dst_gpu]);
                    }
                    if (dst_gpu == world_rank())
                    {
                        ncclRecv(d_A_recv + src_gpu * send_size, sizeof(sa_index_t) * send_size, ncclChar, src_gpu, nccl_comm, context.get_streams(dst_gpu)[src_gpu]);
                    }
                }
            }
            ncclGroupEnd();
            context.sync_all_streams();
            comm_world().barrier();
            hipMemset(d_A_recv, 0, sizeof(sa_index_t) * per_gpu);
        }

        const size_t loop_count = 10;
        std::array<double, loop_count> loop_time;

        for (size_t loop = 0; loop < loop_count; loop++)
        {
            double start = MPI_Wtime();
            ncclGroupStart();
            for (size_t src_gpu = 0; src_gpu < NUM_GPUS; src_gpu++)
            {
                for (size_t dst_gpu = 0; dst_gpu < NUM_GPUS; dst_gpu++)
                {
                    if (src_gpu == world_rank())
                    {
                        ncclSend(d_A_send + dst_gpu * send_size, sizeof(sa_index_t) * send_size, ncclChar, dst_gpu, nccl_comm, context.get_streams(src_gpu)[dst_gpu]);
                    }
                    if (dst_gpu == world_rank())
                    {
                        ncclRecv(d_A_recv + src_gpu * send_size, sizeof(sa_index_t) * send_size, ncclChar, src_gpu, nccl_comm, context.get_streams(dst_gpu)[src_gpu]);
                    }
                }
            }
            ncclGroupEnd();
            context.sync_all_streams();
            comm_world().barrier();
            double end = MPI_Wtime();
            loop_time[loop] = end - start;
            hipMemset(d_A_recv, 0, sizeof(sa_index_t) * per_gpu);
        }
        std::sort(loop_time.begin(), loop_time.end(), std::less_equal<double>());
        double elapsed_time = loop_time[0];
        for (int j = 1; j < loop_count; j++)
        {
            elapsed_time += loop_time[j];
        }
        size_t num_B = sizeof(sa_index_t) * per_gpu;
        size_t B_in_GB = 1 << 30;

        double num_GB = (double)num_B / (double)B_in_GB;
        double avg_time_per_transfer = elapsed_time / ((double)loop_count);
        alg_bandwidth[i - start_offset] = num_GB / avg_time_per_transfer;
        printf("[%lu] Transfer size (B): %10li, Transfer Time Avg|Min|Max (s): %15.9f %15.9f %15.9f, Bandwidth (GB/s): %15.9f\n", world_rank(), num_B, avg_time_per_transfer, loop_time.front(), loop_time.back(), alg_bandwidth[i - start_offset]);
        comm_world().barrier();
        hipFree(d_A_send);
        hipFree(d_A_recv);
        free(A);
        comm_world().barrier();
    }
    if (world_rank() == 0)
    {
        std::ofstream outFile("ncclBandwidthAllToAll8", std::ios::binary);
        if (!outFile)
        {
            std::cerr << "Write Error" << std::endl;
            return;
        }

        outFile.write(reinterpret_cast<char*>(alg_bandwidth.data()), rounds * sizeof(double));
        outFile.close();
    }
}

void alltoallMeasure(MultiGPUContext<NUM_GPUS>& context)
{
    using namespace kamping;
    std::random_device rd;
    std::mt19937 g(rd());
    std::uniform_int_distribution<std::mt19937::result_type> randomDist(0, UINT32_MAX);
    const int rounds = 28;
    const int start_offset = 0;
    std::array<double, rounds> alg_bandwidth;
    for (int iter = 0; iter < rounds; iter++)
    {
        printf("[%lu] iter: %d\n", world_rank(), iter);
        MultiSplit<NUM_GPUS> multi_split(context);
        All2All<NUM_GPUS> all2all(context);
        std::array<sa_index_t*, NUM_GPUS> d_A_send;
        std::array<void*, NUM_GPUS> temp_buffer;
        std::array<sa_index_t*, NUM_GPUS> d_A_recv;
        size_t N = (NUM_GPUS * NUM_GPUS) << iter;

        // Allocate memory for A on CPU
        sa_index_t* A = (sa_index_t*)malloc(N * sizeof(sa_index_t));
        size_t per_gpu = (N / NUM_GPUS);
        if (world_rank() == 0)
        {

            // Initialize all elements of A to random values
            for (size_t j = 0; j < N; j++)
            {
                A[j] = randomDist(g);
            }

            for (size_t gpu_index = 1; gpu_index < NUM_GPUS; gpu_index++)
            {
                comm_world().send(send_buf(std::span<sa_index_t>(A + gpu_index * per_gpu, per_gpu)), send_count(per_gpu), tag(gpu_index), destination(gpu_index));
            }
        }
        else
        {
            comm_world().recv(recv_buf(std::span<sa_index_t>(A + world_rank() * per_gpu, per_gpu)), recv_count(per_gpu), tag(world_rank()), source(0));
        }

        std::array<size_t, NUM_GPUS> temp_storages;
        {
            size_t gpu_index = world_rank();
            // hipStream_t stream = context.get_gpu_default_stream(i);
            sa_index_t* d_A, * d_A_rec;
            hipMalloc(&d_A, per_gpu * sizeof(sa_index_t));
            CUERR;
            d_A_send[gpu_index] = d_A;
            hipMemcpy(d_A_send[gpu_index], A + per_gpu * gpu_index, per_gpu * sizeof(sa_index_t), hipMemcpyHostToDevice);
            CUERR;

            hipMalloc(&d_A_rec, per_gpu * sizeof(sa_index_t));
            CUERR;
            d_A_recv[gpu_index] = d_A_rec;
            // hipcub::DeviceRadixSort::SortKeys(nullptr, temp_storages[gpu_index],
            //     d_A_send[gpu_index], d_A_recv[gpu_index], per_gpu);
            // void* temp;
            // temp_storages[gpu_index] = std::max(temp_storages[gpu_index], 1024ul);
            // temp_storages[gpu_index] = std::max(temp_storages[gpu_index], ((size_t)per_gpu) * sizeof(sa_index_t)) * 4;
            // hipMalloc(&temp, temp_storages[gpu_index]);
            // CUERR;
            // temp_buffer[gpu_index] = temp;
            // hipMemset(temp_buffer[gpu_index], 0, temp_storages[gpu_index]);
            // CUERR;

            hipMemset(d_A_recv[gpu_index], 0, per_gpu * sizeof(sa_index_t));
            CUERR;
        }
        comm_world().barrier();

        hipIpcMemHandle_t handleSend;
        hipIpcGetMemHandle(&handleSend, d_A_send[world_rank()]);
        hipIpcMemHandle_t handleRecv;
        hipIpcGetMemHandle(&handleRecv, d_A_recv[world_rank()]);
        for (size_t dst = 0; dst < NUM_GPUS; dst++) {
            if (context.get_peer_status(world_rank(), dst) != 1) {
                continue;
            }
            comm_world().isend(send_buf(std::span<hipIpcMemHandle_t>(&handleSend, 1)), send_count(1), tag(0), destination(dst));
            comm_world().isend(send_buf(std::span<hipIpcMemHandle_t>(&handleRecv, 1)), send_count(1), tag(1), destination(dst));
        }
        for (size_t src = 0; src < NUM_GPUS; src++) {
            if (context.get_peer_status(world_rank(), src) != 1) {
                continue;
            }
            hipIpcMemHandle_t other_handleSend;
            hipIpcMemHandle_t other_handleRecv;
            comm_world().recv(recv_buf(std::span<hipIpcMemHandle_t>(&other_handleSend, 1)), recv_count(1), tag(0), source(src));
            comm_world().recv(recv_buf(std::span<hipIpcMemHandle_t>(&other_handleRecv, 1)), recv_count(1), tag(1), source(src));
            void* ptrHandleSend;
            void* ptrHandleRecv;
            hipIpcOpenMemHandle(&ptrHandleSend, other_handleSend, hipIpcMemLazyEnablePeerAccess);
            CUERR;
            hipIpcOpenMemHandle(&ptrHandleRecv, other_handleRecv, hipIpcMemLazyEnablePeerAccess);
            CUERR;

            printf("[%lu] opened mem handles from %d\n", world_rank(), src);
            d_A_send[src] = reinterpret_cast<sa_index_t*>(ptrHandleSend);
            d_A_recv[src] = reinterpret_cast<sa_index_t*>(ptrHandleRecv);
        }

        context.sync_default_streams();
        comm_world().barrier();

        std::array<All2AllNodeInfoT<sa_index_t, sa_index_t, sa_index_t>, NUM_GPUS> all2all_node_info;

        split_table_tt<sa_index_t, NUM_GPUS> split_table;
        // std::array<MultiSplitNodeInfoT<sa_index_t, sa_index_t, sa_index_t>, NUM_GPUS> multi_split_node_info;
        // std::array<sa_index_t, NUM_GPUS> dest_lens, src_lens;
        // for (uint gpu_index = 0; gpu_index < NUM_GPUS; ++gpu_index)
        // {
        //     multi_split_node_info[gpu_index].src_keys = d_A_send[gpu_index];
        //     multi_split_node_info[gpu_index].src_values = d_A_send[gpu_index];
        //     multi_split_node_info[gpu_index].src_len = per_gpu;

        //     multi_split_node_info[gpu_index].dest_keys = d_A_recv[gpu_index];
        //     multi_split_node_info[gpu_index].dest_values = d_A_recv[gpu_index];
        //     multi_split_node_info[gpu_index].dest_len = per_gpu;
        //     if (world_rank() == gpu_index)
        //     {
        //         context.get_device_temp_allocator(gpu_index).init(temp_buffer[gpu_index], temp_storages[gpu_index]);
        //     }
        // }

        // PartitioningFunctor<sa_index_t> f(per_gpu, NUM_GPUS - 1);
        // multi_split.execAsync(multi_split_node_info, split_table, src_lens, dest_lens, f);
        size_t send_size = per_gpu / NUM_GPUS;
        for (uint src = 0; src < NUM_GPUS; ++src)
        {
            for (uint dst = 0; dst < NUM_GPUS; ++dst)
            {
                split_table[src][dst] = send_size;
            }
        }

        context.sync_default_streams();

        comm_world().barrier();
        context.sync_all_streams();

        // Warm-up loop
        for (int j = 0; j < 1; j++)
        {
            for (uint gpu_index = 0; gpu_index < NUM_GPUS; ++gpu_index)
            {
                all2all_node_info[gpu_index].src_keys = d_A_send[gpu_index];
                all2all_node_info[gpu_index].src_values = d_A_send[gpu_index];
                all2all_node_info[gpu_index].src_len = per_gpu;

                all2all_node_info[gpu_index].dest_keys = d_A_recv[gpu_index];
                all2all_node_info[gpu_index].dest_values = d_A_recv[gpu_index];
                all2all_node_info[gpu_index].dest_len = per_gpu;
            }
            all2all.execAsync(all2all_node_info, split_table);
            context.sync_all_streams();
            hipMemset(d_A_recv[world_rank()], 0, sizeof(sa_index_t) * per_gpu);
            comm_world().barrier();
        }
        context.sync_all_streams();
        comm_world().barrier();

        // Time ping-pong for loop_count iterations of data transfer size 8*N bytes
        const int loop_count = 10;
        std::array<double, loop_count> loop_time;
        for (int j = 0; j < loop_count; j++)
        {
            double start = MPI_Wtime();
            for (uint gpu_index = 0; gpu_index < NUM_GPUS; ++gpu_index)
            {
                all2all_node_info[gpu_index].src_keys = d_A_send[gpu_index];
                all2all_node_info[gpu_index].src_values = d_A_send[gpu_index];
                all2all_node_info[gpu_index].src_len = per_gpu;

                all2all_node_info[gpu_index].dest_keys = d_A_recv[gpu_index];
                all2all_node_info[gpu_index].dest_values = d_A_recv[gpu_index];
                all2all_node_info[gpu_index].dest_len = per_gpu;
            }
            all2all.execAsync(all2all_node_info, split_table);
            context.sync_all_streams();
            comm_world().barrier();
            double end = MPI_Wtime();
            hipMemset(d_A_recv[world_rank()], 0, sizeof(sa_index_t) * per_gpu);
            loop_time[j] = end - start;
        }

        std::sort(loop_time.begin(), loop_time.end(), std::less_equal<double>());
        double elapsed_time = loop_time[0];
        for (int j = 1; j < loop_count; j++)
        {
            elapsed_time += loop_time[j];
        }
        size_t num_B = sizeof(sa_index_t) * per_gpu;
        size_t B_in_GB = 1 << 30;

        double num_GB = (double)num_B / (double)B_in_GB;
        double avg_time_per_transfer = elapsed_time / ((double)loop_count);
        alg_bandwidth[iter - start_offset] = num_GB / avg_time_per_transfer;
        printf("[%lu] Transfer size (B): %10li, Transfer Time Avg|Min|Max (s): %15.9f %15.9f %15.9f, Bandwidth (GB/s): %15.9f\n", world_rank(), num_B, avg_time_per_transfer, loop_time.front(), loop_time.back(), alg_bandwidth[iter - start_offset]);
        // comm_world().barrier();
        // hipMemcpy(A, d_A_send[world_rank()], per_gpu * sizeof(sa_index_t), hipMemcpyDeviceToHost);
        // CUERR;
        // std::sort(A, A + per_gpu, std::less<sa_index_t>());
        // for(sa_index_t j = 0; j < per_gpu; j++){
        //     if(A[j]-per_gpu*world_rank() != j){
        //         printf("[%lu] A[%u] %u wrong\n", world_rank(), j, A[j]);
        //         break;
        //     }
        // }
        comm_world().barrier();

        hipFree(d_A_send[world_rank()]);
        hipFree(d_A_recv[world_rank()]);
        // hipFree(temp_buffer[gpu_index]);
        free(A);
        comm_world().barrier();
    }

    if (world_rank() == 0)
    {
        std::ofstream outFile("algoBandwidth8", std::ios::binary);
        if (!outFile)
        {
            std::cerr << "Write Error" << std::endl;
            return;
        }

        outFile.write(reinterpret_cast<char*>(alg_bandwidth.data()), rounds * sizeof(double));
        outFile.close();
    }
        }

void warm_up_nccl(MultiGPUContext<NUM_GPUS>& context) {
    ncclComm_t nccl_comm = context.get_nccl();
    std::random_device rd;
    std::mt19937 g(rd());
    std::uniform_int_distribution<std::mt19937::result_type> randomDist(0, INT_MAX);
    int WARM_UP_ROUNDS = 10;

    for (int i = 0; i < WARM_UP_ROUNDS; i++)
    {

        std::vector<int> data(1000);
        for (auto& v : data)
        {
            v = randomDist(g);
        }
        thrust::host_vector<int> h_send(data.begin(), data.end());
        thrust::device_vector<int> send = h_send;
        thrust::device_vector<int> recv;
        recv.reserve(NUM_GPUS * send.size());
        NCCLCHECK(ncclGroupStart());
        for (int dst = 0; dst < NUM_GPUS; dst++)
        {
            NCCLCHECK(ncclSend(thrust::raw_pointer_cast(send.data()), sizeof(int) * send.size(), ncclChar, dst, nccl_comm, context.get_streams(world_rank())[dst]));
        }
        for (size_t src = 0; src < NUM_GPUS; src++)
        {
            NCCLCHECK(ncclRecv(thrust::raw_pointer_cast(recv.data()) + send.size() * src, sizeof(int) * send.size(), ncclChar, src, nccl_comm, context.get_streams(world_rank())[src]));
        }
        NCCLCHECK(ncclGroupEnd());
        context.sync_all_streams();
        comm_world().barrier();
    }
}

int main(int argc, char** argv)
{
    using namespace kamping;
    kamping::Environment e;
    Communicator comm;
    ncclComm_t nccl_comm;
    ncclUniqueId Id;
    int devices;
    hipGetDeviceCount(&devices);
    printf("[%lu] device count: %d\n", world_rank(), devices);
    if (devices == 0)
    {
        printf("[%lu] No GPU found\n", world_rank());
        return 0;
    }
    hipSetDevice(world_rank() % (size_t)devices);
    CUERR;

    if (world_rank() == 0)
    {
        NCCLCHECK(ncclGetUniqueId(&Id));
        comm_world().bcast_single(send_recv_buf(Id));
    }
    else
    {
        Id = comm_world().bcast_single<ncclUniqueId>();
    }

    NCCLCHECK(ncclCommInitRank(&nccl_comm, world_size(), Id, world_rank()));
    printf("[%lu] Active nccl comm\n", world_rank());

    if (argc != 3)
    {
        error("Usage: sa-test <ofile> <ifile> !");
    }

    // for (int i = 0; i < 2; i++)
    // {

    comm_world().barrier();
    char* input = nullptr;



    size_t realLen = 0;
    size_t maxLength = size_t(1024 * 1024) * size_t(1024 * NUM_GPUS);
    size_t inputLen = read_file_into_host_memory(&input, argv[2], realLen, sizeof(sa_index_t), maxLength, NUM_GPUS, 0);
    comm.barrier();
    CUERR;

#ifdef DGX1_TOPOLOGY
    //    const std::array<uint, NUM_GPUS> gpu_ids { 0, 3, 2, 1,  5, 6, 7, 4 };
    //    const std::array<uint, NUM_GPUS> gpu_ids { 1, 2, 3, 0,    4, 7, 6, 5 };
    //    const std::array<uint, NUM_GPUS> gpu_ids { 3, 2, 1, 0,    4, 5, 6, 7 };
    const std::array<uint, NUM_GPUS> gpu_ids{ 3, 2, 1, 0, 4, 7, 6, 5 };

    MultiGPUContext<NUM_GPUS> context(&gpu_ids);
#else
    const std::array<uint, NUM_GPUS> gpu_ids2{ 0, 1, 2, 3 };

    MultiGPUContext<NUM_GPUS> context(nccl_comm, &gpu_ids2, 4);
    warm_up_nccl(context);
    // alltoallMeasure(context);
    // ncclMeasure(context);
    // return 0;
#endif
    SuffixSorter sorter(context, realLen, input);

    sorter.alloc();
    // auto stringPath = ((std::string)argv[3]);
    // int pos = stringPath.find_last_of("/\\");
    // auto fileName = (pos == std::string::npos) ? argv[3] : stringPath.substr(pos + 1);

    // auto& t = kamping::measurements::timer();
    // t.synchronize_and_start(fileName);
    nvtxRangePush("SuffixArray");
    sorter.do_sa();
    nvtxRangePop();
    // t.stop();
    // if (world_rank() == 0)
    //     write_array(argv[2], sorter.get_result(), realLen);

    sorter.done();

    if (world_rank() == 0)
    {
        sorter.print_pd_stats();
        sorter.get_perf_measurements().print(argv[1]);
    }

    hipHostFree(input);
    CUERR;
    // }
    // std::ofstream outFile(argv[1], std::ios::app);
    // t.aggregate_and_print(
    //     kamping::measurements::SimpleJsonPrinter{ outFile, {} });
    // std::cout << std::endl;
    // t.aggregate_and_print(kamping::measurements::FlatPrinter{});
    // std::cout << std::endl;
    return 0;
}
