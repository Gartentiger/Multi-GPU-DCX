#include <iostream>
#include <hip/hip_runtime.h>

int main()
{
    int device_count;
    hipGetDeviceCount(&device_count);

    if (device_count < 2)
    {
        std::cerr << "Mindestens zwei GPUs sind erforderlich!" << std::endl;
        return 1;
    }

    // Prüfen, ob Peer Access zwischen GPU 0 und GPU 1 möglich ist
    int canAccessPeer01 = 0, canAccessPeer10 = 0;
    hipDeviceCanAccessPeer(&canAccessPeer01, 0, 1);
    hipDeviceCanAccessPeer(&canAccessPeer10, 1, 0);

    if (!canAccessPeer01 || !canAccessPeer10)
    {
        std::cerr << "Peer-to-peer Zugriff zwischen GPU 0 und 1 nicht möglich." << std::endl;
        return 1;
    }

    // Peer Access aktivieren
    hipSetDevice(0);
    hipDeviceEnablePeerAccess(1, 0);

    hipSetDevice(1);
    hipDeviceEnablePeerAccess(0, 0);

    std::cout << "Peer Access zwischen GPU 0 und GPU 1 aktiviert.\n";

    // Speicher auf GPU 0 und GPU 1 allokieren
    int *d_gpu0 = nullptr, *d_gpu1 = nullptr;
    hipSetDevice(0);
    hipMalloc(&d_gpu0, sizeof(int));
    int h_value = 42;
    hipMemcpy(d_gpu0, &h_value, sizeof(int), hipMemcpyHostToDevice);

    hipSetDevice(1);
    hipMalloc(&d_gpu1, sizeof(int));

    // Peer-to-peer Copy von GPU 0 nach GPU 1
    hipMemcpyPeer(d_gpu1, 1, d_gpu0, 0, sizeof(int));

    // Wert zurück zur CPU kopieren zum Überprüfen
    int result = 0;
    hipMemcpy(&result, d_gpu1, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Wert von GPU 0 → GPU 1 übertragen: " << result << std::endl;

    // Aufräumen
    hipSetDevice(0);
    hipFree(d_gpu0);

    hipSetDevice(1);
    hipFree(d_gpu1);

    return 0;
}
