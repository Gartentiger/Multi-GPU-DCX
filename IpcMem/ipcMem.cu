#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <mpi.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <random>
#include <ctime>
#include <span>

#include <kamping/checking_casts.hpp>
#include <kamping/collectives/alltoall.hpp>
#include <kamping/collectives/allgather.hpp>
#include <kamping/data_buffer.hpp> 
#include <kamping/environment.hpp>
#include <kamping/measurements/printer.hpp>
#include <kamping/measurements/timer.hpp>
#include <kamping/named_parameters.hpp>
#include <kamping/communicator.hpp>
#include <kamping/p2p/recv.hpp>
#include <kamping/p2p/send.hpp>
#include <kamping/request_pool.hpp>

#define CUDACHECK(cmd) do {                         \
  hipError_t err = cmd;                            \
  if (err != hipSuccess) {                         \
    printf("Failed: Cuda error %s:%d '%s'\n",       \
        __FILE__,__LINE__,hipGetErrorString(err)); \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t res = cmd;                           \
  if (res != ncclSuccess) {                         \
    printf("Failed, NCCL error %s:%d '%s'\n",       \
        __FILE__,__LINE__,ncclGetErrorString(res)); \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

__global__ void printArray(size_t* a, size_t length, size_t rank) {
    if (length >= 64) {
        for (int i = 0; i < 64; i++) {
            printf("[%lu] A[%d]:%lf\n", rank, i, a[i]);
        }
    }
    else {
        for (int i = 0; i < length; i++) {
            printf("[%lu] A[%d]:%lf\n", rank, i, a[i]);
        }
    }
}

const size_t NUM_GPUS = 4;

int main(int argc, char** argv)
{
    using namespace kamping;
    kamping::Environment e;
    Communicator comm;

    int deviceCount;
    CUDACHECK(hipGetDeviceCount(&deviceCount));
    int deviceId = world_rank() % deviceCount;
    std::cout << "Device Id: " << deviceId << std::endl;
    CUDACHECK(hipSetDevice(deviceId));
    std::array<hipStream_t, NUM_GPUS> streams;
    for (size_t i = 0; i < deviceCount; i++)
    {
        CUDACHECK(hipStreamCreate(&streams[i]));
        if (world_rank() == i) {
            continue;
        }
        int canAccess;
        hipDeviceCanAccessPeer(&canAccess, world_rank(), i);
        if (canAccess) {
            CUDACHECK(hipDeviceEnablePeerAccess(i, 0));
            printf("[%lu] peer to [%lu] enabled\n", world_rank(), i);
        }
    }

    size_t N = size_t(1 << 30);

    // Allocate memory for A on CPU
    size_t* A = (size_t*)malloc(N * sizeof(size_t));

    // Initialize all elements of A
    for (int i = 0; i < N; i++) {
        A[i] = i + N * world_rank();
    }
    std::random_device rd;
    std::mt19937 g(rd());
    std::shuffle(A, &A[N - 1], g);
    size_t* d_A;
    CUDACHECK(hipMalloc(&d_A, N * sizeof(size_t)));
    CUDACHECK(hipMemcpy(d_A, A, N * sizeof(size_t), hipMemcpyHostToDevice));


    hipIpcMemHandle_t ownHandle;
    CUDACHECK(hipIpcGetMemHandle(&ownHandle, d_A));
    std::array<hipIpcMemHandle_t, NUM_GPUS> handles;
    std::array<size_t*, NUM_GPUS> pointer;
    comm_world().allgather(send_buf(std::span<hipIpcMemHandle_t>(&ownHandle, 1)), recv_buf(handles));
    printf("[%lu] received handle\n", world_rank());
    for (size_t i = 0; i < NUM_GPUS; i++)
    {
        if (world_rank() == i) {
            pointer[i] = d_A;
            continue;
        }
        void* rawothersd_A;
        hipIpcOpenMemHandle(&rawothersd_A, handles[i], hipIpcMemLazyEnablePeerAccess);
        size_t* other_d_A = reinterpret_cast<size_t*>(rawothersd_A);
        pointer[i] = other_d_A;
    }
    printf("[%lu] opened handles\n", world_rank());
    comm_world().barrier();

    // for (size_t i = 0; i < N; i++)
    // {
    //     //size_t dest = A[i]
    //     CUDACHECK(hipMemcpyPeerAsync(pointer[j] + world_rank() * size_t(N / NUM_GPUS), j, pointer[world_rank()] + world_rank() * size_t(N / NUM_GPUS), world_rank(), sizeof(size_t) * size_t(N / NUM_GPUS), streams[j]));
    // }
    for (size_t j = 0; j < NUM_GPUS; j++)
    {
        CUDACHECK(hipMemcpyPeerAsync(pointer[j] + world_rank() * size_t(N / NUM_GPUS), j, pointer[world_rank()] + world_rank() * size_t(N / NUM_GPUS), world_rank(), sizeof(size_t) * size_t(N / NUM_GPUS), streams[j]));
    }

    for (auto stream : streams)
    {
        CUDACHECK(hipStreamSynchronize(stream));
    }
    comm_world().barrier();
    printArray << <1, 1, 0, streams[0] >> > (pointer[world_rank()], N, world_rank());
    CUDACHECK(hipStreamSynchronize(streams[0]));


    comm_world().barrier();
    for (size_t i = 0; i < NUM_GPUS; i++)
    {
        if (world_rank() == i) {
            continue;
        }
        CUDACHECK(hipIpcCloseMemHandle(pointer[i]));
    }
    comm_world().barrier();
    CUDACHECK(hipFree(d_A));
    return 0;
}
