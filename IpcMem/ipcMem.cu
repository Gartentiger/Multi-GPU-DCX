#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <mpi.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <random>
#include <ctime>
#include <span>

#include <kamping/checking_casts.hpp>
#include <kamping/collectives/alltoall.hpp>
#include <kamping/collectives/allgather.hpp>
#include <kamping/data_buffer.hpp> 
#include <kamping/environment.hpp>
#include <kamping/measurements/printer.hpp>
#include <kamping/measurements/timer.hpp>
#include <kamping/named_parameters.hpp>
#include <kamping/communicator.hpp>
#include <kamping/p2p/recv.hpp>
#include <kamping/p2p/send.hpp>
#include <kamping/request_pool.hpp>

#define CUDACHECK(cmd) do {                         \
  hipError_t err = cmd;                            \
  if (err != hipSuccess) {                         \
    printf("Failed: Cuda error %s:%d '%s'\n",       \
        __FILE__,__LINE__,hipGetErrorString(err)); \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t res = cmd;                           \
  if (res != ncclSuccess) {                         \
    printf("Failed, NCCL error %s:%d '%s'\n",       \
        __FILE__,__LINE__,ncclGetErrorString(res)); \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

__global__ void printArray(double* a, size_t length, size_t rank) {
    for (int i = 0; i < length; i++) {
        printf("[%lu] A[%d]:%lf\n", rank, i, a[i]);
    }
}

const size_t NUM_GPUS = 4;

int main(int argc, char** argv)
{
    using namespace kamping;
    kamping::Environment e;
    Communicator comm;
    int deviceCount;

    CUDACHECK(hipGetDeviceCount(&deviceCount));
    int deviceId = world_rank() % deviceCount;
    std::cout << "Device Id: " << deviceId << std::endl;
    CUDACHECK(hipSetDevice(deviceId));
    std::array<hipStream_t, NUM_GPUS> streams;
    for (size_t i = 0; i < deviceCount; i++)
    {
        CUDACHECK(hipStreamCreate(&streams[i]));
        if (world_rank() == i) {
            continue;
        }
        int canAccess;
        hipDeviceCanAccessPeer(&canAccess, world_rank(), i);
        if (canAccess) {
            CUDACHECK(hipDeviceEnablePeerAccess(world_rank(), i));
            printf("[%lu] peer to [%lu] enabled\n", world_rank(), i);
        }
    }

    long int N = 1 << 6;

    // Allocate memory for A on CPU
    double* A = (double*)malloc(N * sizeof(double));

    // Initialize all elements of A
    for (int i = 0; i < N; i++) {
        A[i] = world_rank() * 100 + i;
    }

    double* d_A;
    CUDACHECK(hipMalloc(&d_A, N * sizeof(double)));
    CUDACHECK(hipMemcpy(d_A, A, N * sizeof(double), hipMemcpyHostToDevice));


    hipIpcMemHandle_t ownHandle;
    CUDACHECK(hipIpcGetMemHandle(&ownHandle, d_A));
    std::array<hipIpcMemHandle_t, NUM_GPUS> handles;
    std::array<double*, NUM_GPUS> pointer;
    comm_world().allgather(send_buf(std::span<hipIpcMemHandle_t>(&ownHandle, 1)), recv_buf(handles));
    printf("[%lu] received handle\n", world_rank());
    for (size_t i = 0; i < NUM_GPUS; i++)
    {
        if (world_rank() == i) {
            pointer[i] = d_A;
            continue;
        }
        void* rawothersd_A;
        hipIpcOpenMemHandle(&rawothersd_A, handles[i], hipIpcMemLazyEnablePeerAccess);
        double* other_d_A = reinterpret_cast<double*>(rawothersd_A);
        pointer[i] = other_d_A;
    }
    printf("[%lu] opened handles\n", world_rank());
    comm_world().barrier();

    for (size_t i = 0; i < NUM_GPUS; i++)
    {
        for (size_t j = 0; j < NUM_GPUS; j++)
        {
            CUDACHECK(hipMemcpyPeerAsync(pointer[j] + i * size_t(N / NUM_GPUS), j, pointer[i] + i * size_t(N / NUM_GPUS), i, sizeof(double) * size_t(N / NUM_GPUS), streams[j]));
        }
    }
    for (auto stream : streams)
    {
        CUDACHECK(hipStreamSynchronize(stream));
    }
    comm_world().barrier();
    printArray << <1, 1, 0, streams[0] >> > (pointer[world_rank()], N, world_rank());
    CUDACHECK(hipStreamSynchronize(streams[0]));


    comm_world().barrier();
    for (size_t i = 0; i < NUM_GPUS; i++)
    {
        if (world_rank() == i) {
            continue;
        }
        CUDACHECK(hipIpcCloseMemHandle(pointer[i]));
    }
    comm_world().barrier();
    CUDACHECK(hipFree(d_A));
    return 0;
}
