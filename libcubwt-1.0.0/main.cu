#include "hip/hip_runtime.h"
#include <stdio.h>
#include "libcubwt.cuh"

// CUDA Kernel-Funktion
__global__ void helloFromGPU()
{
    printf("Hello World from GPU thread %d!\n", threadIdx.x);
}

int main()
{
    printf("Hello World from CPU!\n");
    void *deviceStorage;
    int64_t a = libcubwt_allocate_device_storage(&deviceStorage, 20);
    if (a == LIBCUBWT_NO_ERROR)
    {
        char text[] = "yabbadabbado";
        uint64_t len = strlen(text);

        uint8_t bytes[12]; // oder malloc, wenn dynamisch

        // Kopieren
        for (size_t i = 0; i < len; i++)
        {
            bytes[i] = (uint8_t)text[i];
        }
        uint32_t *isa = nullptr;
        libcubwt_isa(deviceStorage, bytes, isa, len);
        for (int i = 0; i < 12; i++)
        {
            printf("ISA: %u", *isa++);
        }
    }
    // Starte den Kernel mit 1 Block und 5 Threads
    helloFromGPU<<<1, 5>>>();

    // Warten, bis alle GPU-Aufgaben abgeschlossen sind
    hipDeviceSynchronize();

    return 0;
}